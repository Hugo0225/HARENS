#include "hip/hip_runtime.h"
#include "RedundancyEliminator_CUDA.h"

#define CharArraySize(array) strlen(array)

__device__ void SetResultElement(ulong* subResult, uint idx, ulong resultPoint) {
	subResult[idx] = resultPoint;
}

__device__ ulong* GetSubResult(ulong* result, uint blockNum) {
	return &(result[blockNum * THREAD_PER_BLOCK]);
}

__device__ char* GetSubStr(char *str, uint blockNum) {
	return &(str[blockNum * THREAD_PER_BLOCK]);
}

__device__ uint GetUIntFromStr(char* strs, uint idx) {
	return (strs[idx] << 24) | (strs[idx + 1] << 16) | (strs[idx + 2] << 8) | (strs[idx + 3]);
}

__device__ ulong GetULongFromStr(char* strs, uint idx) {
	/*ulong result;
	memcpy((void*)&result, strs, BYTES_IN_ULONG);
	return result;
	*/
	
	ulong result = strs[idx];
	for (int i = 1; i < 8; ++i)
	result = (result << 8) | strs[idx + i];
	return result;
	
}

__device__ char GetChar(char* subStr, uint idx) {
	return subStr[idx];
}

__global__ void Hash(const ulong *TA, const ulong *TB, const ulong *TC, const ulong * TD,
	char *str, const uint windowsNum, ulong *result/*, int *debug*/) {
	if (blockDim.x * blockIdx.x + threadIdx.x >= windowsNum)
		return;

	uint blockNum = blockIdx.x;
	char* subStr = GetSubStr(str, blockNum);
	ulong* subResult = GetSubResult(result, blockNum);

	__shared__ char s_str[THREAD_PER_BLOCK + 3];
	__shared__ char s_str_shift[THREAD_PER_BLOCK + 7];
	uint threadNum = threadIdx.x;

	//This will cover all the elements
	s_str[threadNum] = subStr[threadNum];
	if (threadNum >= THREAD_PER_BLOCK - 3)
		s_str[threadNum + 3] = subStr[threadNum + 3];

	if(threadNum < 7)
		s_str_shift[threadNum] = subStr[threadNum + 4];
	s_str_shift[threadNum + 7] = subStr[threadNum + 11];
	// before starting the computation, make sure all shared memory are set
	__syncthreads();

	int h, i, j, k;

	h = s_str[threadNum];
	i = s_str[threadNum + 1];
	j = s_str[threadNum + 2];
	k = s_str[threadNum + 3];

	ulong resultPoint = GetULongFromStr(s_str_shift, threadNum);
	resultPoint ^= TA[h] ^ TB[i] ^ TC[j] ^ TD[k];
	
	SetResultElement(subResult, threadNum, resultPoint);
	//debug
	/*debug[threadNum * 2] = windowStart;
	debug[threadNum * 2 + 1] = result[windowStart];*/
}

RedundancyEliminator_CUDA::RedundancyEliminator_CUDA(Type type) {
	if (type == NonMultifingerprint)
		circHash = new CircularHash(MAX_CHUNK_NUM);
	else
		circHash = new CircularHashPool(MAX_CHUNK_NUM);
	hashFunc = RabinHash();
	int tableSize = RabinHash::TABLE_ROW_NUM * BYTES_IN_ULONG;
	hipMalloc((void**)&kernelTA, tableSize);
	hipMalloc((void**)&kernelTB, tableSize);
	hipMalloc((void**)&kernelTC, tableSize);
	hipMalloc((void**)&kernelTD, tableSize);
	hipMemcpy(kernelTA, hashFunc.GetTALONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTB, hashFunc.GetTBLONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTC, hashFunc.GetTCLONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTD, hashFunc.GetTDLONG(), tableSize, hipMemcpyHostToDevice);
	//The real software need to generate a initial file named 0xFF here
	//Check Circular.cpp to see the reason
}

void RedundancyEliminator_CUDA::SetupRedundancyEliminator_CUDA(Type type) {
	if (type == NonMultifingerprint)
		circHash = new CircularHash(MAX_CHUNK_NUM);
	else
		circHash = new CircularHashPool(MAX_CHUNK_NUM);
	hashFunc = RabinHash();
	int tableSize = RabinHash::TABLE_ROW_NUM * BYTES_IN_ULONG;
	hipMalloc((void**)&kernelTA, tableSize);
	hipMalloc((void**)&kernelTB, tableSize);
	hipMalloc((void**)&kernelTC, tableSize);
	hipMalloc((void**)&kernelTD, tableSize);
	hipMemcpy(kernelTA, hashFunc.GetTALONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTB, hashFunc.GetTBLONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTC, hashFunc.GetTCLONG(), tableSize, hipMemcpyHostToDevice);
	hipMemcpy(kernelTD, hashFunc.GetTDLONG(), tableSize, hipMemcpyHostToDevice);
	//The real software need to generate a initial file named 0xFF here
	//Check Circular.cpp to see the reason
}

RedundancyEliminator_CUDA::~RedundancyEliminator_CUDA() {
	delete circHash;
	hipFree(kernelTA);
	hipFree(kernelTB);
	hipFree(kernelTC);
	hipFree(kernelTD);
	//The real software would delete all the generated files here
}

/*
Add a new chunck into the file system, if the hash value queue is full, also delete the oldest chunk.
*/
void RedundancyEliminator_CUDA::addNewChunk(ulong hashValue, char* chunk, uint chunkSize, bool isDuplicate) {
	ulong to_be_del = circHash->Add(hashValue, isDuplicate);
	/*fstream file(hashValue.c_str(), std::fstream::in | std::fstream::out);
	file << chunk;
	file.close();*/
}

void RedundancyEliminator_CUDA::ChunkHashing(uint* indices, int indicesNum, char* package, 
	char** chunkList, ulong* chunkHashValueList, uint* chunkLenList) {
	uint prevIdx = 0;
	for (int i = 0; i < indicesNum; ++i) {
		if (prevIdx == 0) {
			prevIdx = indices[i];
			continue;
		}
		chunkLenList[i - 1] = indices[i] - prevIdx;
		chunkList[i - 1] = &(package[prevIdx]);

		//Mind! never use sizeof(chunk) to check the chunk size
		chunkHashValueList[i - 1] = computeChunkHash(chunkList[i - 1], chunkLenList[i - 1]);
		prevIdx = indices[i];
	}
}

uint RedundancyEliminator_CUDA::ChunkMatching(deque<ulong> &hashValues, deque<tuple<char*, uint>> &chunks) {
	uint duplicationSize = 0;
	bool isDuplicate;
	deque<ulong>::const_iterator hashValueIter = hashValues.begin();
	deque<tuple<char*, uint>>::const_iterator chunksIter = chunks.begin();
	while (hashValueIter != hashValues.end()) {
		if (circHash->Find(*hashValueIter)) {
			duplicationSize += get<1>(*chunksIter);
			isDuplicate = true;
		}
		else {
			isDuplicate = false;
		}
		addNewChunk(*hashValueIter, get<0>(*chunksIter), get<1>(*chunksIter), isDuplicate);
		++hashValueIter;
		++chunksIter;
	}
	return duplicationSize;
}

void RedundancyEliminator_CUDA::ChunkHashingAscynWithCircularQueue(uint* indices, int indicesNum, char* package,
	CircularPairQueue<ulong, uint> &chunkHashQ) {
	//uint duplicationSize = 0;
	uint prevIdx = 0;
	char* chunk;
	uint chunkLen;
	ulong chunkHashValue;
	for (int i = 0; i < indicesNum; ++i) {
		if (prevIdx == 0) {
			prevIdx = indices[i];
			continue;
		}
		chunk = &(package[prevIdx]);
		chunkLen = indices[i] - prevIdx;
		chunkHashValue = computeChunkHash(chunk, chunkLen);
		chunkHashQ.Push(chunkHashValue, chunkLen);

		//Mind! never use sizeof(chunk) to check the chunk size
		prevIdx = indices[i];
	}
}

void RedundancyEliminator_CUDA::ChunkHashingAscyn(uint* indices, int indicesNum, char* package, 
	ulong* chunkHashValueList, uint* chunkLenList, mutex &chunkMutex) {
	//uint duplicationSize = 0;
	uint prevIdx = 0;
	char* chunk;
	for (int i = 0; i < indicesNum; ++i) {
		if (prevIdx == 0) {
			prevIdx = indices[i];
			continue;
		}
		chunkLenList[i - 1] = indices[i] - prevIdx;
		chunk = &(package[prevIdx]);

		//Mind! never use sizeof(chunk) to check the chunk size
		chunkMutex.lock();
		chunkHashValueList[i - 1] = computeChunkHash(chunk, chunkLenList[i - 1]);
		chunkMutex.unlock();
		prevIdx = indices[i];
	}
}

uint RedundancyEliminator_CUDA::fingerPrinting(deque<uint> indexQ, char* package) {
	/*deque<uchar*> hashValues;
	deque<tuple<char*, uint>> chunks;
	ChunkHashing(indexQ, package, hashValues, chunks);
	return ChunkMatching(hashValues, chunks);*/
	uint duplicationSize = 0;
	uint prevIdx = 0;
	char* chunk;
	ulong chunkHash;
	uint chunkLen;
	bool isDuplicate;
	for (deque<uint>::const_iterator iter = indexQ.begin(); iter != indexQ.end(); ++iter) {
		if (prevIdx == 0) {
			prevIdx = *iter;
			continue;
		}
		chunkLen = *iter - prevIdx;
		chunk = &(package[prevIdx]);

		//Mind! never use sizeof(chunk) to check the chunk size
		chunkHash = computeChunkHash(chunk, chunkLen);
		if (circHash->Find(chunkHash)) { //find duplications
			duplicationSize += chunkLen;
			isDuplicate = true;
		}
		else {
			isDuplicate = false;
		}
		addNewChunk(chunkHash, chunk, chunkLen, isDuplicate);
		prevIdx = *iter;
	}
	//system("pause");
	return duplicationSize;
}

void RedundancyEliminator_CUDA::RabinHashAsync(char* inputKernel, char* inputHost, uint inputLen, ulong* resultKernel, ulong* resultHost, hipStream_t stream) {
	hipMemcpyAsync(inputKernel, inputHost,	inputLen, hipMemcpyHostToDevice, stream);
	Hash << <BLOCK_NUM, THREAD_PER_BLOCK, 0, stream>> > (kernelTA, kernelTB, kernelTC, kernelTD,
		inputKernel, (inputLen - WINDOW_SIZE + 1), resultKernel/*, debugDevice*/);
	hipMemcpyAsync(resultHost, resultKernel,
		(inputLen - WINDOW_SIZE + 1) * BYTES_IN_ULONG, hipMemcpyDeviceToHost, stream);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR1: %s \n", hipGetErrorString(error));
	}
}

//take a kernel global memory address and the size as input
uint RedundancyEliminator_CUDA::eliminateRedundancy(char* package, uint packageSize) {
	/*char *kernelInput;
	hipMalloc((void**)&kernelInput, MAX_KERNEL_INPUT_LEN);*/
	uint totalDuplicationSize = 0;
	deque<uint> indexQ;
	char* packageInput[2];
	char* kernelInput[2];
	ulong* resultHost[2];
	ulong *kernelResult[2];
	clock_t start;
	clock_t end;
	double time = 0;

	const uint MAX_WINDOW_NUM = MAX_KERNEL_INPUT_LEN - WINDOW_SIZE + 1;
	hipHostMalloc((void**)&packageInput[0], MAX_KERNEL_INPUT_LEN);
	hipHostMalloc((void**)&packageInput[1], MAX_KERNEL_INPUT_LEN);
	hipHostMalloc((void**)&resultHost[0], MAX_WINDOW_NUM * BYTES_IN_ULONG);
	hipHostMalloc((void**)&resultHost[1], MAX_WINDOW_NUM * BYTES_IN_ULONG);

	hipMalloc((void**)&kernelInput[0], MAX_KERNEL_INPUT_LEN);
	hipMalloc((void**)&kernelInput[1], MAX_KERNEL_INPUT_LEN);
	hipMalloc((void**)&kernelResult[0], MAX_WINDOW_NUM * BYTES_IN_ULONG);
	hipMalloc((void**)&kernelResult[1], MAX_WINDOW_NUM * BYTES_IN_ULONG);

	hipStream_t* streams = new hipStream_t[2];
	for (int i = 0; i < 2; ++i)
		hipStreamCreate(&(streams[i]));

	int bufferIdx = 0;
	uint curInputLen = MAX_KERNEL_INPUT_LEN, curWindowNum, curFilePos = 0;
	for (int iterator = 0; curInputLen == MAX_KERNEL_INPUT_LEN; ++iterator) {
		curInputLen = min(MAX_KERNEL_INPUT_LEN, packageSize - curFilePos);
		curWindowNum = curInputLen - WINDOW_SIZE + 1;
		memcpy(packageInput[bufferIdx], &(package[curFilePos]), curInputLen);

		start = clock();
		hipStreamSynchronize(streams[bufferIdx]);
		
		//Because of unblock cuda process, deal with the 2 iteration eariler cuda output here
		if (iterator > 1) {
			for (uint j = 0; j < MAX_WINDOW_NUM; ++j) {
				if ((resultHost[bufferIdx][j] & P_MINUS) == 0) { // marker found
					indexQ.push_back(j);
				}
			}
			end = clock();
			time += (end - start) * 1000 / CLOCKS_PER_SEC;
			totalDuplicationSize += fingerPrinting(indexQ, &(package[curFilePos - (MAX_WINDOW_NUM << 1)]));
			indexQ.clear();
		}

		RabinHashAsync(kernelInput[bufferIdx], packageInput[bufferIdx], curInputLen, kernelResult[bufferIdx], resultHost[bufferIdx], streams[bufferIdx]);

		bufferIdx ^= 1;
		curFilePos += curWindowNum;
	}

	start = clock();
	hipDeviceSynchronize();
	for (uint j = 0; j < MAX_WINDOW_NUM; ++j) {
		if ((resultHost[bufferIdx][j] & P_MINUS) == 0) { // marker found
			indexQ.push_back(j);
		}
	}
	end = clock();
	time += (end - start) * 1000 / CLOCKS_PER_SEC;
	totalDuplicationSize += fingerPrinting(indexQ, &(package[curFilePos - MAX_WINDOW_NUM - curWindowNum]));
	indexQ.clear();
	start = clock();
	for (uint j = 0; j < curWindowNum; ++j) {
		if ((resultHost[bufferIdx ^ 1][j] & P_MINUS) == 0) { // marker found
			indexQ.push_back(j);
		}
	}
	end = clock();
	time += (end - start) * 1000 / CLOCKS_PER_SEC;
	totalDuplicationSize += fingerPrinting(indexQ, &(package[curFilePos - curWindowNum]));

	printf("chunking time: %f ms\n", time);
	hipFree(kernelResult[0]);
	hipHostFree(resultHost[0]);
	hipFree(kernelInput[0]);
	hipHostFree(packageInput[0]);
	hipFree(kernelResult[1]);
	hipHostFree(resultHost[1]);
	hipFree(kernelInput[1]);
	hipHostFree(packageInput[1]);
	//hipFree(kernelInput);
	return totalDuplicationSize;
}

/*
Compute the hash value of chunk, should use sha256 to avoid collision
*/
inline ulong RedundancyEliminator_CUDA::computeChunkHash(char* chunk, uint chunkSize) {
	return hashFunc.Hash(chunk, chunkSize);
	//SHA((uchar*)chunk, chunkSize, hashValue);
}
